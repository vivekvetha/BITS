
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>

#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(1); \
        } \
    } while(0)

__global__ void add_vectors(double *a, double *b, double *c, int n) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id < n) {
        c[id] = a[id] + b[id];
    }
}

void cpu_vector_add(const std::vector<double>& a, const std::vector<double>& b, std::vector<double>& c, int n) {
    for (int i = 0; i < n; ++i) {
        c[i] = a[i] + b[i];
    }
}

void benchmark_vector_size(int N) {
    std::cout << "\n" << std::string(60, '=') << std::endl;
    std::cout << "BENCHMARKING VECTOR SIZE: " << N << " elements (" << (N * sizeof(double)) / (1024*1024) << " MB)" << std::endl;
    std::cout << std::string(60, '=') << std::endl;

    size_t bytes = N * sizeof(double);

    std::vector<double> h_A(N);
    std::vector<double> h_B(N);
    std::vector<double> h_C(N, 0.0);
    std::vector<double> h_C_cpu(N);

    // Initialize input vectors
    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    // Allocate GPU memory
    double *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, bytes));
    CHECK_CUDA(hipMalloc(&d_B, bytes));
    CHECK_CUDA(hipMalloc(&d_C, bytes));

    // Copy data to GPU
    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice));

    // Launch configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "Launch config: " << blocksPerGrid << " blocks x " << threadsPerBlock << " threads" << std::endl;

    // Create timing events
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Launch kernel with timing
    CHECK_CUDA(hipEventRecord(start));
    add_vectors<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float gpu_kernel_time_ms = 0;
    CHECK_CUDA(hipEventElapsedTime(&gpu_kernel_time_ms, start, stop));

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost));

    // CPU timing
    auto start_cpu = std::chrono::high_resolution_clock::now();
    cpu_vector_add(h_A, h_B, h_C_cpu, N);
    auto stop_cpu = std::chrono::high_resolution_clock::now();
    auto cpu_time_ms = std::chrono::duration_cast<std::chrono::microseconds>(stop_cpu - start_cpu).count() / 1000.0;

    bool success = true;
    double tolerance = 1.0e-14;

    for (int i = 0; i < std::min(100, N); ++i) {
        if (std::abs(h_C[i] - 3.0) > tolerance) {
            success = false;
            break;
        }
    }
    for (int i = std::max(0, N-100); i < N && success; ++i) {
        if (std::abs(h_C[i] - 3.0) > tolerance) {
            success = false;
            break;
        }
    }

    // Results
    if (success) {
        std::cout << "Vector Addition Successful!" << std::endl;
        std::cout << "CPU Execution Time: " << cpu_time_ms << " ms" << std::endl;
        std::cout << "GPU Kernel Execution Time: " << gpu_kernel_time_ms << " ms" << std::endl;

        if (gpu_kernel_time_ms > 0) {
            double speedup = cpu_time_ms / gpu_kernel_time_ms;
            double throughput = (N / 1e6) / (gpu_kernel_time_ms / 1000.0);
            std::cout << "GPU Speedup: " << speedup << "x" << std::endl;
            std::cout << "GPU Throughput: " << throughput << " M elements/sec" << std::endl;
        }
    } else {
        std::cout << "Vector Addition Failed!" << std::endl;
    }

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
}

int main() {
    // Test multiple vector sizes
    std::vector<int> sizes = {
        1 << 20,
        1 << 22,
        1 << 24,
        1 << 26,
        1 << 28
    };

    for (int size : sizes) {
        benchmark_vector_size(size);
    }

    return 0;
}
